#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cmath>
#include "hip/hip_math_constants.h"

#include <thrust/extrema.h>

using namespace std;

#define N 15
#define PHASE 45
#define BASE (360 / PHASE)

/* size in bytes */
#define BATCH_SIZE 6442450944
#define BATCH BATCH_SIZE / sizeof(float)

__device__ __forceinline__ float atomicMaxFloat(float* addr, float value) {
    float old;
    old = !signbit(value) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) :
        __uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));

    return old;
}

__global__ void kernel(
  float *c,
  unsigned long long offset
  )
{
  __shared__ float signal_Re[BASE];
  __shared__ float signal_Im[BASE];

  if (threadIdx.x < BASE) {
    float rad = 2 * HIP_PI * threadIdx.x / BASE;
    signal_Re[threadIdx.x] = sinf(rad);
    signal_Im[threadIdx.x] = cosf(rad);
  }

  __syncthreads();

  __shared__ char signal[N];

  unsigned long long signal_part = blockIdx.x + offset + 1;

  for (int i = 0; i < threadIdx.x; i++) {
    if (signal_part == 0) { break; }
    signal_part /= BASE;
  }

  signal[threadIdx.x] = signal_part % BASE;

  if (threadIdx.x == 0) {
    return;
  }

  __shared__ float max;
  max = 0;

  float sum_Re = 0;
  float sum_Im = 0;
  for (int i = 0; i + threadIdx.x < N; i++) {
    int idx = (BASE + signal[i] - signal[threadIdx.x + i]) % BASE;

    sum_Re += signal_Re[idx];
    sum_Im += signal_Im[idx];
  }

  float akf = sqrtf(sum_Re * sum_Re + sum_Im * sum_Im);

  atomicMaxFloat(&max, akf);

  if (threadIdx.x != 1) {
    return;
  }

  c[blockIdx.x] = max;
}

unsigned long long start_kernel(
  unsigned long long offset
  )
{
  float *host_c = (float*)malloc(BATCH_SIZE);
  float host_akf;

  float *dev_c;

  hipMalloc(&dev_c, BATCH_SIZE);
  
  int threadsPerBlock = N;
  unsigned long blocksInGrid = BATCH;

  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventRecord(start, 0);
  kernel<<< blocksInGrid, threadsPerBlock >>>(dev_c, offset);
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float t;

  hipEventElapsedTime(&t, start, stop);
  printf("gpu time: %f\n", t);

  hipEventCreate(&start);
  hipEventRecord(start, 0);
  unsigned long long result = thrust::min_element(thrust::device, dev_c, dev_c + BATCH) - dev_c;
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(&host_akf, dev_c + result, sizeof(float), hipMemcpyDeviceToHost);

  hipEventElapsedTime(&t, start, stop);
  printf("thrust::min_element time: %f\n", t);

  printf("index: %zd\n", result + offset);
  printf("best signal: %zd\n", result + offset + 1);
  printf("akf: %f\n", host_akf);

  free(host_c);
  hipFree(dev_c);

  return 0;
}

unsigned long long get_num_combinations()
{
  unsigned long long num_combinations = BASE;

  for (int i = 0; i < N; i++) {
    num_combinations = num_combinations * BASE;
  }

  return num_combinations;
}

int main()
{
  unsigned long long num_combinations = get_num_combinations();
  size_t size = num_combinations * sizeof(float);

  if (size <= 0) {
    cout << "result array size error" << endl;
    return 1;
  }

  unsigned long num_batches = size / BATCH_SIZE;
  num_batches = num_batches ? num_batches : 1;

  printf("BATCH COUNT: %ld\n", num_batches);

  // unsigned long long result;

  for (unsigned long i = 0; i < num_batches; i++) {
    unsigned long long offset = i * BATCH;

    printf("\n --- BATCH %d --- \n\n", i + 1);

    unsigned long long batch_result = start_kernel(offset);
  }
}