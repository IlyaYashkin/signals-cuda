
#include <hip/hip_runtime.h>
#include <stdint.h>
#include "hip/hip_math_constants.h"

/**
 * Вычисление матрицы перехода
*/

__device__ void getTransitionMatrix(
  float2* transition_matrix,
  uint32_t signal_size,
  uint32_t base
)
{
  for (int i = 0; i < (int) ceilf((float) base / (float) signal_size); i++) {
    int idx = threadIdx.x + i * signal_size;

    if (idx > base - 1) { break; }

    float rad = 2 * HIP_PI * idx / base;

    transition_matrix[idx].x = cosf(rad);
    transition_matrix[idx].y = sinf(rad);
  }
}


/**
 * Генерация сигнала
*/

__device__ void getSignal(
  float2* signal,
  float2* transition_matrix,
  uint64_t offset,
  uint32_t signal_size,
  uint32_t base
)
{
  if (threadIdx.x >= signal_size) { return; }

  uint64_t signal_part = blockIdx.x + offset;

  for (int i = 0; i < threadIdx.x && threadIdx.x < signal_size; i++) {
    if (signal_part == 0) { break; }
    signal_part /= base;
  }

  uint32_t t_idx = signal_part % base;

  signal[threadIdx.x].x = transition_matrix[t_idx].x;
  signal[threadIdx.x].y = transition_matrix[t_idx].y;
}


/**
 * Поиск максимального значения
*/

__device__ void reduceMax(
  float2* signal,
  uint32_t signal_size
)
{
  uint32_t i = blockDim.x / 2;

  while (i != 0) {
    if (threadIdx.x <= i && threadIdx.x + i < signal_size) {
      signal[threadIdx.x].x = fmaxf(signal[threadIdx.x].x, signal[threadIdx.x + i].x);
    }

    __syncthreads();

    i /= 2;
  }
}


/**
 * Поиск АКФ
*/

__device__ float findAkf(
  float2* signal,
  uint32_t signal_size
)
{
  float2 sum = {0.0, 0.0};

  for (int i = 0; i + threadIdx.x < signal_size; i++) {
    sum.x += signal[threadIdx.x + i].x * signal[i].x - signal[threadIdx.x + i].y * -signal[i].y;
    sum.y += signal[threadIdx.x + i].x * -signal[i].y + signal[threadIdx.x + i].y * signal[i].x;
  }

  return sqrtf(sum.x * sum.x + sum.y * sum.y);
}


/**
 * Поиск оптимального сигнала
 * Заполняет массив float c[] максимальными лепестками
*/

__global__ void kernel(
  float *c,
  uint64_t offset,
  uint32_t signal_size,
  uint32_t base
)
{
  extern __shared__ float2 s[];

  float2 *transition_matrix = s;

  getTransitionMatrix(transition_matrix, signal_size, base);

  __syncthreads();

  float2 *signal = (float2*)&s[base];

  getSignal(
    signal,
    transition_matrix,
    offset,
    signal_size,
    base
  );

  __syncthreads();

  float akf = 0;

  if (threadIdx.x != 0) {
    akf = findAkf(signal, signal_size);
  }

  __syncthreads();

  if (threadIdx.x < signal_size) {
    signal[threadIdx.x].x = akf;
  }

  __syncthreads();

  reduceMax(signal, signal_size);

  __syncthreads();

  if (threadIdx.x != 0) {
    return;
  }

  c[blockIdx.x] = signal[0].x;
}


/**
 * Поиск суммы элементов массива
*/

__device__ void reduceSum(float2* arr, uint32_t signal_size)
{
  uint32_t i = blockDim.x / 2;

  while (i != 0) {
    if (threadIdx.x <= i && threadIdx.x + i < signal_size) {
      arr[threadIdx.x].x += arr[threadIdx.x + i].x;
    }

    __syncthreads();

    i /= 2;
  }
}


/**
 * Поиск оптимального сигнала с заданным сдвигом Доплера
*/

__global__ void kernel_doppler(
  float *c,
  uint64_t offset,
  uint32_t signal_size,
  uint32_t base
)
{
  extern __shared__ float2 s[];

  float2 *transition_matrix = s;

  getTransitionMatrix(transition_matrix, signal_size, base);

  __syncthreads();

  float2 *signal = (float2*)&s[base];

  getSignal(signal, transition_matrix, offset, signal_size, base);

  __syncthreads();

  float akf = findAkf(signal, signal_size);

  __syncthreads();

  float2 *akf_arr = (float2*)&s[base + signal_size];

  if (threadIdx.x < signal_size) {
    signal[threadIdx.x].x = akf;
    signal[threadIdx.x].y = threadIdx.x;

    akf_arr[threadIdx.x].x = akf;
    akf_arr[threadIdx.x].y = threadIdx.x;
  }

  __syncthreads();

  reduceMax(signal, signal_size);

  __syncthreads();

  if (threadIdx.x == 0) {
    akf_arr[(int) signal[0].y].x = 0;
  }
  
  __syncthreads();

  reduceSum(akf_arr, signal_size);

  __syncthreads();

  if (threadIdx.x != 0) {
    return;
  }

  c[blockIdx.x] = signal[0].x - akf_arr[0].x;
}
