#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cmath>
#include "hip/hip_math_constants.h"

#include <thrust/extrema.h>

using namespace std;

#define N 5
#define PHASE 180
#define BASE (360 / PHASE)

__device__ __forceinline__ float atomicMaxFloat(float* addr, float value) {
    float old;
    old = !signbit(value) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) :
        __uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));

    return old;
}

__global__ void kernel(float *c, float *signal_Re, float *signal_Im)
{
  __shared__ char signal[N];

  int signal_part = blockIdx.x + 1;

  for (int i = 0; i < threadIdx.x; i++) {
    if (signal_part == 0) { break; }
    signal_part /= BASE;
  }

  signal[threadIdx.x] = signal_part % BASE;

  if (threadIdx.x == 0) {
    return;
  }

  __shared__ float max;
  max = 0.0;

  float sum_Re = 0;
  float sum_Im = 0;
  for (int i = 0; i + threadIdx.x < N; i++) {
    int idx = (BASE + signal[i] - signal[threadIdx.x + i]) % BASE;

    sum_Re += signal_Re[idx];
    sum_Im += signal_Im[idx];
  }

  float akf = sqrtf(sum_Re * sum_Re + sum_Im * sum_Im);

  c[blockDim.x * blockIdx.x + threadIdx.x] = akf;
}

int main()
{
  size_t m_trans_size = BASE * sizeof(float);

  float *signal_Re = (float*)malloc(m_trans_size);
  float *signal_Im = (float*)malloc(m_trans_size);

  float *dev_signal_Re;
  float *dev_signal_Im;

  for (int i = 0; i < BASE; i++) {
    float rad = 2 * HIP_PI * i / BASE;
    signal_Im[i] = sin(rad);
    signal_Re[i] = cos(rad);
  }

  hipMalloc(&dev_signal_Re, m_trans_size);
  hipMalloc(&dev_signal_Im, m_trans_size);
  hipMemcpy(dev_signal_Re, signal_Re, m_trans_size, hipMemcpyHostToDevice);
  hipMemcpy(dev_signal_Im, signal_Im, m_trans_size, hipMemcpyHostToDevice);


  unsigned long num_combinations = pow(BASE, N) - 2;

  size_t size = N * num_combinations * sizeof(float);

  float *host_c = (float*)malloc(size);

  float *dev_c;

  hipMalloc(&dev_c, size);
  
  int threadsPerBlock = N;
  unsigned long blocksInGrid = num_combinations;

  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventRecord(start, 0);
  kernel<<< blocksInGrid, threadsPerBlock >>>(dev_c, dev_signal_Re, dev_signal_Im);
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float t;

  hipEventElapsedTime(&t, start, stop);
  printf("gpu time: %f\n", t);

  hipMemcpy(host_c, dev_c, size, hipMemcpyDeviceToHost);

  int counter = 0;

  for (int i = 0; i < N * num_combinations; i++)
  {
      if (counter == N)
      {
        cout << ' ' << i / N << endl;
        counter = 0;
      }

      printf("%.2f ", host_c[i]);
      counter++;
  }


  free(host_c);
  hipFree(dev_c);

  return 0;
}
